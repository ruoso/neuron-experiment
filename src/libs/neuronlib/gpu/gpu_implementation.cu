#include "hip/hip_runtime.h"
#include "gpu_interface.h"
#include "../neuron.h"
#include <hip/hip_runtime.h>
#include <memory>

namespace neuronlib {
namespace gpu {

// Forward declaration of kernel
__global__ void process_activations_kernel(
    const GpuActivationBatch* input,
    GpuProcessingResults* output
);

// GPU memory management class
class GpuMemoryManager {
private:
    // Device pointers
    GpuActivationBatch* d_input_;
    GpuProcessingResults* d_output_;
    
public:
    GpuMemoryManager() : d_input_(nullptr), d_output_(nullptr) {}
    
    ~GpuMemoryManager() {
        cleanup();
    }
    
    bool initialize() {
        // Allocate GPU memory for input/output structures
        hipError_t err;
        
        err = hipMalloc(&d_input_, sizeof(GpuActivationBatch));
        if (err != hipSuccess) {
            return false;
        }
        
        err = hipMalloc(&d_output_, sizeof(GpuProcessingResults));
        if (err != hipSuccess) {
            return false;
        }
        
        return true;
    }
    
    
    bool process_batch(const GpuActivationBatch& input, GpuProcessingResults& output) {
        hipError_t err;
        
        // Copy input to GPU
        err = hipMemcpy(d_input_, &input, sizeof(GpuActivationBatch), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            return false;
        }
        
        // Clear output structure
        output.clear();
        err = hipMemcpy(d_output_, &output, sizeof(GpuProcessingResults), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            return false;
        }
        
        // Launch kernel
        int block_size = 256;  // Threads per block
        int grid_size = (input.target_count + block_size - 1) / block_size;  // Blocks in grid
        
        
        process_activations_kernel<<<grid_size, block_size>>>(
            d_input_, d_output_
        );
        
        // Check for kernel launch errors
        err = hipGetLastError();
        if (err != hipSuccess) {
            return false;
        }
        
        // Wait for kernel to complete
        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            return false;
        }
        
        // Copy result back to host
        err = hipMemcpy(&output, d_output_, sizeof(GpuProcessingResults), hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            return false;
        }
        
        
        return true;
    }
    
    void cleanup() {
        if (d_input_) { hipFree(d_input_); d_input_ = nullptr; }
        if (d_output_) { hipFree(d_output_); d_output_ = nullptr; }
    }
};

// Global GPU memory manager instance
static std::unique_ptr<GpuMemoryManager> g_gpu_manager = nullptr;

// Initialize GPU system
bool initialize_gpu() {
    if (g_gpu_manager) {
        return true;  // Already initialized
    }
    
    // Check for CUDA-capable devices
    int device_count;
    hipError_t err = hipGetDeviceCount(&device_count);
    if (err != hipSuccess || device_count == 0) {
        return false;
    }
    
    // Get device properties
    hipDeviceProp_t prop;
    err = hipGetDeviceProperties(&prop, 0);
    if (err != hipSuccess) {
        return false;
    }
    
    
    // Create memory manager
    g_gpu_manager = std::make_unique<GpuMemoryManager>();
    if (!g_gpu_manager->initialize()) {
        g_gpu_manager.reset();
        return false;
    }
    
    return true;
}

// Cleanup GPU system
void cleanup_gpu() {
    g_gpu_manager.reset();
}

// Main GPU processing interface function
bool process_activations_gpu(const GpuActivationBatch& input, 
                            GpuProcessingResults& output) {
    if (!g_gpu_manager) {
        return false;
    }
    
    // Process the batch
    return g_gpu_manager->process_batch(input, output);
}

} // namespace gpu
} // namespace neuronlib