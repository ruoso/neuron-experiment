#include "hip/hip_runtime.h"
#include "gpu_interface.h"
#include "../brain.h"
#include "../dendrite.h"
#include <hip/hip_runtime.h>
#include <>
#include <cmath>

namespace neuronlib {
namespace gpu {

// CUDA kernel for processing neural activations
__global__ void process_activations_kernel(
    const GpuActivationBatch* input,
    GpuProcessingResults* output
) {
    int target_idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (target_idx >= input->target_count) {
        return;
    }
    
    uint32_t target_address = input->target_addresses[target_idx];
    uint32_t activation_count = input->activation_counts[target_idx];
    uint32_t activation_offset = input->activation_offsets[target_idx];
    uint32_t current_timestamp = input->current_timestamp;
    uint32_t timing_window = input->timing_window;
    
    // Process activations within timing window
    float total_input = 0.0f;
    float total_absolute_weights = 0.0f;
    
    for (uint32_t i = 0; i < activation_count; ++i) {
        uint32_t act_idx = activation_offset + i;
        
        if (act_idx >= MAX_GPU_ACTIVATIONS) break;
        
        uint32_t timestamp = input->activation_timestamps[act_idx];
        
        // Check if activation is within timing window
        if (current_timestamp >= timestamp && 
            current_timestamp <= timestamp + timing_window) {
            
            float value = input->activation_values[act_idx];
            float weight = input->activation_weights[act_idx];
            
            float weighted_input = value * weight;
            total_input += weighted_input;
            total_absolute_weights += fabsf(weight);
        }
    }
    
    // Normalize output to 0-1 range based on total absolute weights
    if (total_absolute_weights > 0.0f) {
        total_input = (total_input + total_absolute_weights) / (2.0f * total_absolute_weights);
        total_input = fmaxf(0.0f, fminf(1.0f, total_input));
    }
    
    // Check if this is a neuron address (simplified check for GPU)
    if (is_neuron_address(target_address)) {
        uint32_t neuron_index = target_address >> DENDRITE_ADDRESS_BITS;
        
        if (neuron_index < MAX_NEURONS) {
            float threshold = input->target_thresholds[target_idx];
            bool neuron_fired = total_input >= threshold;
            
            // Check refractory period
            constexpr uint32_t REFRACTORY_PERIOD = 5;
            uint32_t last_firing_time = input->target_last_activations[target_idx];
            bool in_refractory = (current_timestamp - last_firing_time) < REFRACTORY_PERIOD;
            
            if (neuron_fired && in_refractory) {
                neuron_fired = false;
            }
            
            // Add soma activation for CPU processing (regardless of firing)
            uint32_t soma_idx = atomicAdd(&output->soma_count, 1);
            if (soma_idx < MAX_GPU_TARGETS) {
                output->soma_neuron_indices[soma_idx] = neuron_index;
                output->soma_activation_levels[soma_idx] = total_input;
                output->soma_target_addresses[soma_idx] = target_address;
            }
            
            // Apply Hebbian learning - weight updates
            constexpr float LEARNING_RATE = 0.01f;
            for (uint32_t i = 0; i < activation_count; ++i) {
                uint32_t act_idx = activation_offset + i;
                if (act_idx >= MAX_GPU_ACTIVATIONS) break;
                
                uint32_t timestamp = input->activation_timestamps[act_idx];
                if (current_timestamp >= timestamp && 
                    current_timestamp <= timestamp + timing_window) {
                    
                    uint32_t source_address = input->source_addresses[act_idx];
                    float value = input->activation_values[act_idx];
                    float weight = input->activation_weights[act_idx];
                    float weighted_input = value * weight;
                    
                    float delta = 0.0f;
                    if (neuron_fired && weighted_input > 0.0f) {
                        // Strengthen weights that contributed to firing
                        delta = LEARNING_RATE * value;
                    } else if (!neuron_fired) {
                        // Slightly weaken weights when neuron doesn't fire
                        delta = -LEARNING_RATE * 0.1f;
                    }
                    
                    if (delta != 0.0f) {
                        uint32_t weight_idx = atomicAdd(&output->weight_update_count, 1);
                        if (weight_idx < MAX_GPU_WEIGHT_UPDATES) {
                            output->weight_addresses[weight_idx] = source_address;
                            output->weight_deltas[weight_idx] = delta;
                        }
                    }
                }
            }
        }
    } else if (is_terminal_address(target_address)) {
        // Terminal: propagate to its branch
        uint32_t branch_address = get_terminal_branch(target_address);
        if (total_input > 0.0f) {
            uint32_t new_act_idx = atomicAdd(&output->new_activation_count, 1);
            if (new_act_idx < MAX_GPU_OUTPUTS) {
                output->new_activations_target[new_act_idx] = branch_address;
                output->new_activations_value[new_act_idx] = total_input;
                output->new_activations_source[new_act_idx] = target_address;
                output->new_activations_timestamp[new_act_idx] = current_timestamp;
            }
        }
    } else if (is_branch_address(target_address)) {
        // Branch: check threshold and either propagate up or fire neuron
        uint32_t parent_address = get_parent_branch(target_address);
        
        // Intermediate branch - propagate to parent
        if (total_input > 0.0f) {
            uint32_t new_act_idx = atomicAdd(&output->new_activation_count, 1);
            if (new_act_idx < MAX_GPU_OUTPUTS) {
                output->new_activations_target[new_act_idx] = parent_address;
                output->new_activations_value[new_act_idx] = total_input;
                output->new_activations_source[new_act_idx] = target_address;
                output->new_activations_timestamp[new_act_idx] = current_timestamp;
            }
        }
    }
}

} // namespace gpu
} // namespace neuronlib